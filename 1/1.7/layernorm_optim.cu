#include <stdio.h>
#include <random>
#include<hip/hip_runtime.h>
#include<hip/hip_bf16.h>

#define NUM_THREADS 256
#define NUM_STREAMS 4
#define B 16384
#define H 4096

#if (B * H) % NUM_STREAMS != 0
#error (B * H) must be divisible by NUM_STREAMS
#endif

#define CHUNK_ROWS (B / NUM_STREAMS)
#define CHUNK_SIZE (CHUNK_ROWS * H)
#define CHUNK_BYTES (CHUNK_SIZE * sizeof(__hip_bfloat16))
// #define CHUNK_BYTES (CHUNK_SIZE * sizeof(float))


// x_i = (x1, x2, ..., xn)
// x_i = (xi - mean) / sqrt(var^2 + eps)
// y_i = gamma * x_i + beta
__launch_bounds__(256)
__global__ void LayerNormKernel(const __hip_bfloat16* __restrict__ input, __hip_bfloat16* __restrict__ output, const __hip_bfloat16* __restrict__ gamma, const __hip_bfloat16* __restrict__ beta, float epsilon) {
    // One block process one row
    int rowIdx = blockIdx.x;
    int tId = threadIdx.x;
    int blockSize = blockDim.x;

    // Shared Memory
    __shared__ __hip_bfloat16 shared_row[H];
    for (int i=tId; i<H; i+=blockDim.x) {
        shared_row[i] = input[rowIdx * H + i];
    }

    // Check shared memory is loaded
    __syncthreads();

    // Declare a shared memory array to save partial_sums
    __shared__ float partial_accum[NUM_THREADS];
    __shared__ float partial_accum_sq[NUM_THREADS];

    float local_accum = 0.0f;
    float local_accum_sq = 0.0f;
    for (int i=tId; i<H; i+=blockSize) {
        float val = __bfloat162float(shared_row[i]);
        local_accum += val;
        local_accum_sq += val * val;
    }
    partial_accum[tId] = local_accum;
    partial_accum_sq[tId] = local_accum_sq;
    __syncthreads();

    // This will happen in order
    // offset = 16: threads 0-15 will get the sum of data from threads 16-31
    // offset = 8: threads 0-7 will get data from threads 8-15
    // ...
    // Finally, offset = 1: threads 0 will get data from threads 1
    for (int offset=blockSize/2; offset>0; offset>>=1) {
        if (tId < offset) {
            partial_accum[tId] += partial_accum[tId + offset];
            partial_accum_sq[tId] += partial_accum_sq[tId + offset];
        }
        __syncthreads();
    }

    // Calculate `mean` and `var`
    __shared__ float mean, var;
    if (tId == 0) {
        float accum = partial_accum[0];
        float accum_sq = partial_accum_sq[0];
        mean = accum / H;
        var = (accum_sq / H) - (mean * mean);
    }
    __syncthreads();

    // Calculate `x_i`
    float inv_std = rsqrtf(var + epsilon);
    for (int i=tId; i<H; i+=blockSize) {
        float val = __bfloat162float(shared_row[i]);
        float norm = (val - mean) * inv_std;
        float gamma_val = __bfloat162float(gamma[i]);
        float beta_val = __bfloat162float(beta[i]);

        output[rowIdx * H + i] = __float2bfloat16(gamma_val * norm + beta_val);
    }
}


static void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error %s: %s", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}


float getRand(std::mt19937& gen, std::uniform_real_distribution<float>& dist) {
    return dist(gen);
}


int main() {
    // Init
    int sizeMat = B * H * sizeof(__hip_bfloat16);
    int sizeArr = H * sizeof(__hip_bfloat16);

    float epsilon = 1e-10;
    __hip_bfloat16 *h_input, *h_output, *h_gamma, *h_beta;

    // Malloc host data
    checkCudaError(hipHostMalloc((void**)&h_input, sizeMat, hipHostMallocDefault), "hipHostMalloc h_input");
    checkCudaError(hipHostMalloc((void**)&h_output, sizeMat, hipHostMallocDefault), "hipHostMalloc h_output");
    checkCudaError(hipHostMalloc((void**)&h_gamma, sizeArr, hipHostMallocDefault), "hipHostMalloc h_gamma");
    checkCudaError(hipHostMalloc((void**)&h_beta, sizeArr, hipHostMallocDefault), "hipHostMalloc h_beta");
    
    // Random initialization
    // Init random number generator and distribution
    std::mt19937 gen(std::random_device{}());
    std::uniform_real_distribution<float> dist(-1.5f, 1.5f);

    for (int i=0; i<H; ++i) {
        h_gamma[i] = __float2bfloat16_rn(getRand(gen, dist));
        h_beta[i] = __float2bfloat16_rn(getRand(gen, dist));

        for (int j=0; j<B; ++j) {
            h_input[j * H + i] = __float2bfloat16_rn(getRand(gen, dist));
        }
    }

    // Init device data
    __hip_bfloat16 *d_input, *d_output, *d_gamma, *d_beta;
    checkCudaError(hipMalloc((void**)&d_input, sizeMat), "hipMalloc d_input");
    checkCudaError(hipMalloc((void**)&d_output, sizeMat), "hipMalloc d_output");
    checkCudaError(hipMalloc((void**)&d_gamma, sizeArr), "hipMalloc d_gamma");
    checkCudaError(hipMalloc((void**)&d_beta, sizeArr), "hipMalloc d_beta");

    // Create Streams and Events
    hipStream_t streams[NUM_STREAMS];
    hipEvent_t doneEvents[NUM_STREAMS];
    for (int i=0; i<NUM_STREAMS; ++i) {
        checkCudaError(hipStreamCreate(&streams[i]), "hipStreamCreate");
        checkCudaError(hipEventCreate(&doneEvents[i]), "hipEventCreate");
    }
    hipEvent_t start, stop;
    checkCudaError(hipEventCreate(&start), "hipEventCreate start");
    checkCudaError(hipEventCreate(&stop), "hipEventCreate stop");


    // Copy gamma and beta
    checkCudaError(hipMemcpy(d_gamma, h_gamma, sizeArr, hipMemcpyHostToDevice), "hipMemcpy h_gamma => d_gamma");
    checkCudaError(hipMemcpy(d_beta, h_beta, sizeArr, hipMemcpyHostToDevice), "hipMemcpy h_beta => d_beta");

    // Start to record
    checkCudaError(hipEventRecord(start), "hipEventRecord start");

    // Assign tasks to different streams
    for (int i=0; i<NUM_STREAMS; ++i) {
        int offset = i * CHUNK_SIZE;

        // Copy data from host to device
        checkCudaError(hipMemcpyAsync(d_input + offset, h_input + offset, CHUNK_BYTES, hipMemcpyHostToDevice, streams[i]), "hipMemcpy d_input => h_input");

        // Launch LayerNorm Kernel
        dim3 block(NUM_THREADS);
        dim3 grid(CHUNK_ROWS);
        LayerNormKernel<<<grid, block, 0, streams[i]>>>(d_input + offset, d_output + offset, d_gamma, d_beta, epsilon);
        checkCudaError(hipGetLastError(), "LayerNormKernel launched");

        // Record the doneEvent of stream
        checkCudaError(hipEventRecord(doneEvents[i], streams[i]), "hipEventRecord doneEvent");
    }
    
    // Sync all operations
    for (int i = 0; i < NUM_STREAMS; ++i) {
        checkCudaError(hipEventSynchronize(doneEvents[i]), "hipEventSynchronize doneEvent");
    }

    // Stop to record
    checkCudaError(hipEventRecord(stop), "hipEventRecord stop");
    checkCudaError(hipEventSynchronize(stop), "hipEventSynchronize stop");

    // Print timing
    float ms = 0.0f;
    checkCudaError(hipEventElapsedTime(&ms, start, stop), "hipEventElapsedTime");
    printf("Optimal Kernel Spend Time: %f ms\n", ms);

    // Destroy
    checkCudaError(hipHostFree(h_input), "hipFree h_input");
    checkCudaError(hipHostFree(h_output), "hipFree h_output");
    checkCudaError(hipHostFree(h_gamma), "hipFree h_gamma");
    checkCudaError(hipHostFree(h_beta), "hipFree h_beta");

    for (int i=0; i<NUM_STREAMS; ++i) {
        checkCudaError(hipStreamDestroy(streams[i]), "hipStreamDestroy streams");
        checkCudaError(hipEventDestroy(doneEvents[i]), "hipEventDestroy events");        
    }
    checkCudaError(hipEventDestroy(start), "hipEventDestroy start");
    checkCudaError(hipEventDestroy(stop), "hipEventDestroy stop");
    checkCudaError(hipFree(d_input), "hipFree d_input");
    checkCudaError(hipFree(d_output), "hipFree d_output");
    checkCudaError(hipFree(d_gamma), "hipFree d_gamma");
    checkCudaError(hipFree(d_beta), "hipFree d_beta");

    return 0;
}