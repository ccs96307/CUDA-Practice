#include <stdio.h>
#include <random>
#include<hip/hip_runtime.h>

#define NUM_THREADS 256
#define NUM_STREAMS 2
#define B 16384
#define H 4096

#if (B * H) % NUM_STREAMS != 0
#error (B * H) must be divisible by NUM_STREAMS
#endif

#define CHUNK_ROWS (B / NUM_STREAMS)
#define CHUNK_SIZE (CHUNK_ROWS * H)
#define CHUNK_BYTES (CHUNK_SIZE * sizeof(float))

// x_i = (x1, x2, ..., xn)
// x_i = (xi - mean) / sqrt(var^2 + eps)
// y_i = gamma * x_i + beta
__global__ void LayerNormKernel(const float* __restrict__ input, float* __restrict__ output, const float* __restrict__ gamma, const float* __restrict__ beta, float epsilon) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    float accum = 0.0f;
    float mean = 0.0f;
    float var = 0.0f;
    
    if (idx >= B / NUM_STREAMS) {
        return;
    }
    
    // Calculate `mean`
    for (int i=0; i<H; ++i) {
        accum += input[idx * H + i];
    }
    mean = accum / H;
    accum = 0;

    // Calculate `var^2`
    for (int i=0; i<H; ++i) {
        float diff = input[idx * H + i] - mean;
        accum += diff * diff;
    }
    var = accum / H;

    // Calculate `x_i`
    for (int i=0; i<H; ++i) {
        output[idx * H + i] = gamma[i] * ((input[idx * H + i] - mean) / sqrtf(var + epsilon)) + beta[i];
    }
}


static void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error %s: %s", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}


float getRand(std::mt19937& gen, std::uniform_real_distribution<float>& dist) {
    return dist(gen);
}


int main() {
    // Init
    int sizeMat = B * H * sizeof(float);
    int sizeArr = H * sizeof(float);

    float epsilon = 1e-10;
    float *h_input, *h_output, *h_gamma, *h_beta;

    // Malloc host data
    checkCudaError(hipHostMalloc((void**)&h_input, sizeMat, hipHostMallocDefault), "hipHostMalloc h_input");
    checkCudaError(hipHostMalloc((void**)&h_output, sizeMat, hipHostMallocDefault), "hipHostMalloc h_output");
    checkCudaError(hipHostMalloc((void**)&h_gamma, sizeArr, hipHostMallocDefault), "hipHostMalloc h_gamma");
    checkCudaError(hipHostMalloc((void**)&h_beta, sizeArr, hipHostMallocDefault), "hipHostMalloc h_beta");
    
    // Random initialization
    // Init random number generator and distribution
    std::mt19937 gen(std::random_device{}());
    std::uniform_real_distribution<float> dist(-1.5f, 1.5f);

    for (int i=0; i<H; ++i) {
        h_gamma[i] = getRand(gen, dist);
        h_beta[i] = getRand(gen, dist);

        for (int j=0; j<B; ++j) {
            h_input[j * H + i] = getRand(gen, dist);
        }
    }

    // Init device data
    float *d_input, *d_output, *d_gamma, *d_beta;
    checkCudaError(hipMalloc((void**)&d_input, sizeMat), "hipMalloc d_input");
    checkCudaError(hipMalloc((void**)&d_output, sizeMat), "hipMalloc d_output");
    checkCudaError(hipMalloc((void**)&d_gamma, sizeArr), "hipMalloc d_gamma");
    checkCudaError(hipMalloc((void**)&d_beta, sizeArr), "hipMalloc d_beta");

    // Create Streams and Events
    hipStream_t streams[NUM_STREAMS];
    for (int i=0; i<NUM_STREAMS; ++i) {
        checkCudaError(hipStreamCreate(&streams[i]), "hipStreamCreate");
    }
    hipEvent_t start, stop;
    checkCudaError(hipEventCreate(&start), "hipEventCreate start");
    checkCudaError(hipEventCreate(&stop), "hipEventCreate stop");  

    // Copy gamma and beta
    checkCudaError(hipMemcpy(d_gamma, h_gamma, sizeArr, hipMemcpyHostToDevice), "hipMemcpy h_gamma => d_gamma");
    checkCudaError(hipMemcpy(d_beta, h_beta, sizeArr, hipMemcpyHostToDevice), "hipMemcpy h_beta => d_beta");

    // Start to record
    checkCudaError(hipEventRecord(start), "hipEventRecord start");

    // Assign tasks to different streams
    for (int i=0; i<NUM_STREAMS; ++i) {
        int offset = i * CHUNK_SIZE;

        // Copy data from host to device
        checkCudaError(hipMemcpyAsync(d_input + offset, h_input + offset, CHUNK_BYTES, hipMemcpyHostToDevice, streams[i]), "hipMemcpy d_input => h_input");

        // Launch LayerNorm Kernel
        dim3 block(NUM_THREADS);
        dim3 grid((CHUNK_ROWS + block.x - 1) / block.x);
        LayerNormKernel<<<grid, block, 0, streams[i]>>>(d_input + offset, d_output + offset, d_gamma, d_beta, epsilon);
        checkCudaError(hipGetLastError(), "LayerNormKernel launched");
    }
    
    // Sync all operations
    checkCudaError(hipDeviceSynchronize(), "hipDeviceSynchronize for timing");

    // Stop to record
    checkCudaError(hipEventRecord(stop), "hipEventRecord stop");
    checkCudaError(hipEventSynchronize(stop), "hipEventSynchronize stop");

    // Print timing
    float ms = 0.0f;
    checkCudaError(hipEventElapsedTime(&ms, start, stop), "hipEventElapsedTime");
    printf("Original Kernel Spend Time: %f ms\n", ms);

    // Destroy
    checkCudaError(hipHostFree(h_input), "hipFree h_input");
    checkCudaError(hipHostFree(h_output), "hipFree h_output");
    checkCudaError(hipHostFree(h_gamma), "hipFree h_gamma");
    checkCudaError(hipHostFree(h_beta), "hipFree h_beta");

    for (int i=0; i<NUM_STREAMS; ++i) {
        checkCudaError(hipStreamDestroy(streams[i]), "hipStreamDestroy streams");
    }
    checkCudaError(hipEventDestroy(start), "hipEventDestroy start");
    checkCudaError(hipEventDestroy(stop), "hipEventDestroy stop");
    checkCudaError(hipFree(d_input), "hipFree d_input");
    checkCudaError(hipFree(d_output), "hipFree d_output");
    checkCudaError(hipFree(d_gamma), "hipFree d_gamma");
    checkCudaError(hipFree(d_beta), "hipFree d_beta");

    return 0;
}