#include <stdio.h>
#include <random>
#include<hip/hip_runtime.h>

#define NUM_THREADS 256
#define B 16384
#define H 4096

// x_i = (x1, x2, ..., xn)
// x_i = (xi - mean) / sqrt(var^2 + eps)
// y_i = gamma * x_i + beta
__global__ void LayerNormKernel(const float* __restrict__ input, float* __restrict__ output, const float* __restrict__ gamma, const float* __restrict__ beta, float epsilon) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    float accum = 0.0f;
    float mean = 0.0f;
    float var = 0.0f;
    
    if (idx >= B) {
        return;
    }
    
    // Calculate `mean`
    for (int i=0; i<H; ++i) {
        accum += input[idx * H + i];
    }
    mean = accum / H;
    accum = 0;

    // Calculate `var^2`
    for (int i=0; i<H; ++i) {
        float diff = input[idx * H + i] - mean;
        accum += diff * diff;
    }
    var = accum / H;

    // Calculate `x_i`
    for (int i=0; i<H; ++i) {
        output[idx * H + i] = gamma[i] * ((input[idx * H + i] - mean) / sqrtf(var + epsilon)) + beta[i];
    }
}


static void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error %s: %s", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}


float getRand(std::mt19937& gen, std::uniform_real_distribution<float>& dist) {
    return dist(gen);
}


int main() {
    // Init
    int sizeMat = B * H * sizeof(float);
    int sizeArr = H * sizeof(float);

    float epsilon = 1e-10;
    float *h_input, *h_output, *h_gamma, *h_beta;

    // Malloc host data
    checkCudaError(hipHostMalloc((void**)&h_input, sizeMat, hipHostMallocDefault), "hipHostMalloc h_input");
    checkCudaError(hipHostMalloc((void**)&h_output, sizeMat, hipHostMallocDefault), "hipHostMalloc h_output");
    checkCudaError(hipHostMalloc((void**)&h_gamma, sizeArr, hipHostMallocDefault), "hipHostMalloc h_gamma");
    checkCudaError(hipHostMalloc((void**)&h_beta, sizeArr, hipHostMallocDefault), "hipHostMalloc h_beta");
    
    // Random initialization
    // Init random number generator and distribution
    std::mt19937 gen(std::random_device{}());
    std::uniform_real_distribution<float> dist(-1.5f, 1.5f);

    for (int i=0; i<H; ++i) {
        h_gamma[i] = getRand(gen, dist);
        h_beta[i] = getRand(gen, dist);

        for (int j=0; j<B; ++j) {
            h_input[j * H + i] = getRand(gen, dist);
        }
    }

    // Init device data
    float *d_input, *d_output, *d_gamma, *d_beta;
    checkCudaError(hipMalloc((void**)&d_input, sizeMat), "hipMalloc d_input");
    checkCudaError(hipMalloc((void**)&d_output, sizeMat), "hipMalloc d_output");
    checkCudaError(hipMalloc((void**)&d_gamma, sizeArr), "hipMalloc d_gamma");
    checkCudaError(hipMalloc((void**)&d_beta, sizeArr), "hipMalloc d_beta");

    // Create event
    hipEvent_t start, stop;
    checkCudaError(hipEventCreate(&start), "hipEventCreate start");
    checkCudaError(hipEventCreate(&stop), "hipEventCreate stop");  

    // Start to record
    checkCudaError(hipEventRecord(start), "hipEventRecord start");

    // Copy data from host to device
    checkCudaError(hipMemcpyAsync(d_input, h_input, sizeMat, hipMemcpyHostToDevice), "hipMemcpy h_input => d_input");
    checkCudaError(hipMemcpyAsync(d_gamma, h_gamma, sizeArr, hipMemcpyHostToDevice), "hipMemcpy h_gamma => d_gamma");
    checkCudaError(hipMemcpyAsync(d_beta, h_beta, sizeArr, hipMemcpyHostToDevice), "hipMemcpy h_beta => d_beta");

    // Launch LayerNorm Kernel
    dim3 block(NUM_THREADS);
    dim3 grid((B + block.x - 1) / block.x);
    LayerNormKernel<<<grid, block>>>(d_input, d_output, d_gamma, d_beta, epsilon);
    checkCudaError(hipGetLastError(), "LayerNormKernel launched");

    // Sync all operations
    checkCudaError(hipDeviceSynchronize(), "hipDeviceSynchronize for timing");

    // Stop to record
    checkCudaError(hipEventRecord(stop), "hipEventRecord stop");
    checkCudaError(hipEventSynchronize(stop), "hipEventSynchronize stop");

    // Print timing
    float ms = 0.0f;
    checkCudaError(hipEventElapsedTime(&ms, start, stop), "hipEventElapsedTime");
    printf("Original Kernel Spend Time: %f ms\n", ms);

    // Destroy
    checkCudaError(hipHostFree(h_input), "hipFree h_input");
    checkCudaError(hipHostFree(h_output), "hipFree h_output");
    checkCudaError(hipHostFree(h_gamma), "hipFree h_gamma");
    checkCudaError(hipHostFree(h_beta), "hipFree h_beta");

    checkCudaError(hipEventDestroy(start), "hipEventDestroy start");
    checkCudaError(hipEventDestroy(stop), "hipEventDestroy stop");
    checkCudaError(hipFree(d_input), "hipFree d_input");
    checkCudaError(hipFree(d_output), "hipFree d_output");
    checkCudaError(hipFree(d_gamma), "hipFree d_gamma");
    checkCudaError(hipFree(d_beta), "hipFree d_beta");

    return 0;
}