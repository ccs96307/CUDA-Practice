#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 10000000
#define NUM_THREADS 256
#define NUM_STREAMS 4

#if N % NUM_STREAMS != 0
#error N must be divisible by NUM_STREAMS
#endif

#define CHUNK_SIZE (N / NUM_STREAMS)
#define CHUNK_BYTES (CHUNK_SIZE * sizeof(float))
#define TOTAL_DATA_SIZE (N * sizeof(float))


// The compute formula: y = scale * relu(x + b)
__global__ void add_relu_scaleKernel(const float* x, const float *b, float scale, float* y, int chunkSize) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < chunkSize) {
        y[idx] = scale * (fmaxf(0.0f, x[idx] + b[idx]));
    }
}


static void checkCudaError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}


int main() {
    // Init host data (try to use fixed memory)
    float scale = 2.0f;
    float *h_x, *h_b, *h_y;
    checkCudaError(hipHostMalloc((void**)&h_x, TOTAL_DATA_SIZE, hipHostMallocDefault), "hipHostMalloc h_x");
    checkCudaError(hipHostMalloc((void**)&h_b, TOTAL_DATA_SIZE, hipHostMallocDefault), "hipHostMalloc h_b");
    checkCudaError(hipHostMalloc((void**)&h_y, TOTAL_DATA_SIZE, hipHostMallocDefault), "hipHostMalloc h_y");    

    for (int i = 0; i < N; ++i) {
        h_x[i] = static_cast<float>(i);
        h_b[i] = static_cast<float>(i);
    }

    // Init device data
    float *d_x, *d_b, *d_y;
    checkCudaError(hipMalloc((void**)&d_x, TOTAL_DATA_SIZE), "hipMalloc d_x");
    checkCudaError(hipMalloc((void**)&d_b, TOTAL_DATA_SIZE), "hipMalloc d_b");
    checkCudaError(hipMalloc((void**)&d_y, TOTAL_DATA_SIZE), "hipMalloc d_y");

    // Create Streams and Events
    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; ++i) {
        checkCudaError(hipStreamCreate(&streams[i]), "hipStreamCreate");
    }
    hipEvent_t start, stop;
    checkCudaError(hipEventCreate(&start), "hipEventCreate start");
    checkCudaError(hipEventCreate(&stop), "hipEventCreate stop");

    // Start to record
    checkCudaError(hipEventRecord(start), "hipEventRecord start");

    // Assign tasks to different streams
    for (int i=0; i<NUM_STREAMS; ++i) {
        int offset = i * CHUNK_SIZE;

        // Copy data from host to device
        checkCudaError(hipMemcpyAsync(d_x + offset, h_x + offset, CHUNK_BYTES, hipMemcpyHostToDevice, streams[i]), "hipMemcpy h_x => d_x");
        checkCudaError(hipMemcpyAsync(d_b + offset, h_b + offset, CHUNK_BYTES, hipMemcpyHostToDevice, streams[i]), "hipMemcpy h_b => d_b");

        // Init block and grid
        dim3 block(NUM_THREADS);
        dim3 grid((CHUNK_SIZE + block.x - 1) / block.x);

        // Launch Add Kernel
        add_relu_scaleKernel<<<grid, block, 0, streams[i]>>>(
            d_x + offset,
            d_b + offset,
            scale,
            d_y + offset,
            CHUNK_SIZE
        );
        checkCudaError(hipGetLastError(), "add_relu_scaleKernel launched");
        checkCudaError(hipMemcpyAsync(h_y + offset, d_y + offset, CHUNK_BYTES, hipMemcpyDeviceToHost, streams[i]), "hipMemcpy d_y => h_y");
    }

    // Sync all operations
    checkCudaError(hipDeviceSynchronize(), "hipDeviceSynchronize for timing");

    // Stop to record
    checkCudaError(hipEventRecord(stop), "hipEventRecord stop");
    checkCudaError(hipEventSynchronize(stop), "hipEventSynchronize stop");

    // Print the time
    float ms = 0;
    checkCudaError(hipEventElapsedTime(&ms, start, stop), "hipEventElapsedTime");
    printf("GPU Kernel Spend Time with %d streams: %f ms\n", NUM_STREAMS, ms);

    // Destory
    checkCudaError(hipHostFree(h_x), "hipFree h_x");
    checkCudaError(hipHostFree(h_b), "hipFree h_b");
    checkCudaError(hipHostFree(h_y), "hipFree h_y");

    for (int i=0; i<NUM_STREAMS; ++i) {
        checkCudaError(hipStreamDestroy(streams[i]), "hipStreamDestroy streams");
    }
    checkCudaError(hipEventDestroy(start), "hipEventDestroy start");
    checkCudaError(hipEventDestroy(stop), "hipEventDestroy stop");
    checkCudaError(hipFree(d_x), "hipFree d_x");
    checkCudaError(hipFree(d_b), "hipFree d_b");
    checkCudaError(hipFree(d_y), "hipFree d_y");

    return 0;
}