#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 10000000
#define NUM_THREADS 256
#define NUM_STREAMS 4

#if N % NUM_STREAMS != 0
#error N must be divisible by NUM_STREAMS
#endif

#define CHUNK_SIZE (N / NUM_STREAMS)
#define CHUNK_BYTES (CHUNK_SIZE * sizeof(float))
#define TOTAL_DATA_SIZE (N * sizeof(float))


// The compute formula: y = scale * relu(x + b)
__global__ void addKernel(const float* x, const float *b, float *tmp1, int chunkSize) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < chunkSize) {
        tmp1[idx] = x[idx] + b[idx];
    }
}


__global__ void reluKernel(const float* tmp1, float* tmp2, int chunkSize) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < chunkSize) {
        tmp2[idx] = fmaxf(0.0f, tmp1[idx]);
    }
}


__global__ void scaleKernel(const float* tmp2, float* y, float scale, int chunkSize) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < chunkSize) {
        y[idx] = tmp2[idx] * scale;
    }
}


static void checkCudaError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}


int main() {
    // Init host data (try to use fixed memory)
    float scale = 2.0f;
    float *h_x, *h_b, *h_tmp1, *h_tmp2, *h_y;
    checkCudaError(hipHostMalloc((void**)&h_x, TOTAL_DATA_SIZE, hipHostMallocDefault), "hipHostMalloc h_x");
    checkCudaError(hipHostMalloc((void**)&h_b, TOTAL_DATA_SIZE, hipHostMallocDefault), "hipHostMalloc h_b");
    checkCudaError(hipHostMalloc((void**)&h_tmp1, TOTAL_DATA_SIZE, hipHostMallocDefault), "hipHostMalloc h_tmp1");
    checkCudaError(hipHostMalloc((void**)&h_tmp2, TOTAL_DATA_SIZE, hipHostMallocDefault), "hipHostMalloc h_tmp2");
    checkCudaError(hipHostMalloc((void**)&h_y, TOTAL_DATA_SIZE, hipHostMallocDefault), "hipHostMalloc h_y");    

    for (int i = 0; i < N; ++i) {
        h_x[i] = static_cast<float>(i);
        h_b[i] = static_cast<float>(i);
    }

    // Init device data
    float *d_x, *d_b, *d_tmp1, *d_tmp2, *d_y;
    checkCudaError(hipMalloc((void**)&d_x, TOTAL_DATA_SIZE), "hipMalloc d_x");
    checkCudaError(hipMalloc((void**)&d_b, TOTAL_DATA_SIZE), "hipMalloc d_b");
    checkCudaError(hipMalloc((void**)&d_tmp1, TOTAL_DATA_SIZE), "hipMalloc d_tmp1");
    checkCudaError(hipMalloc((void**)&d_tmp2, TOTAL_DATA_SIZE), "hipMalloc d_tmp2");
    checkCudaError(hipMalloc((void**)&d_y, TOTAL_DATA_SIZE), "hipMalloc d_y");

    // Create Streams and Events
    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; ++i) {
        checkCudaError(hipStreamCreate(&streams[i]), "hipStreamCreate");
    }
    hipEvent_t start, stop;
    checkCudaError(hipEventCreate(&start), "hipEventCreate start");
    checkCudaError(hipEventCreate(&stop), "hipEventCreate stop");

    // Start to record
    checkCudaError(hipEventRecord(start), "hipEventRecord start");

    // Assign tasks to different streams
    for (int i=0; i<NUM_STREAMS; ++i) {
        int offset = i * CHUNK_SIZE;

        // Copy data from host to device
        checkCudaError(hipMemcpyAsync(d_x + offset, h_x + offset, CHUNK_BYTES, hipMemcpyHostToDevice, streams[i]), "hipMemcpy h_x => d_x");
        checkCudaError(hipMemcpyAsync(d_b + offset, h_b + offset, CHUNK_BYTES, hipMemcpyHostToDevice, streams[i]), "hipMemcpy h_b => d_b");

        // Init block and grid
        dim3 block(NUM_THREADS);
        dim3 grid((CHUNK_SIZE + block.x - 1) / block.x);

        // Launch Add Kernel
        addKernel<<<grid, block, 0, streams[i]>>>(
            d_x + offset,
            d_b + offset,
            d_tmp1 + offset,
            CHUNK_SIZE
        );
        checkCudaError(hipGetLastError(), "addKerenl launched");

        // Launch ReLU Kernel
        reluKernel<<<grid, block, 0, streams[i]>>>(
            d_tmp1 + offset,
            d_tmp2 + offset,
            CHUNK_SIZE
        );
        checkCudaError(hipGetLastError(), "reluKerenl launched");
 
        // Launch ReLU Kernel
        scaleKernel<<<grid, block, 0, streams[i]>>>(
            d_tmp2 + offset,
            d_y + offset,
            scale,
            CHUNK_SIZE
        );
        checkCudaError(hipGetLastError(), "scaleKerenl launched");
        checkCudaError(hipMemcpyAsync(h_y + offset, d_y + offset, CHUNK_BYTES, hipMemcpyDeviceToHost, streams[i]), "hipMemcpy d_y => h_y");
    }

    // Sync all operations
    checkCudaError(hipDeviceSynchronize(), "hipDeviceSynchronize for timing");

    // Stop to record
    checkCudaError(hipEventRecord(stop), "hipEventRecord stop");
    checkCudaError(hipEventSynchronize(stop), "hipEventSynchronize stop");

    // Print the time
    float ms = 0;
    checkCudaError(hipEventElapsedTime(&ms, start, stop), "hipEventElapsedTime");
    printf("GPU Kernel Spend Time with %d streams: %f ms\n", NUM_STREAMS, ms);

    // Destory
    checkCudaError(hipHostFree(h_x), "hipFree h_x");
    checkCudaError(hipHostFree(h_b), "hipFree h_b");
    checkCudaError(hipHostFree(h_tmp1), "hipFree h_tmp1");
    checkCudaError(hipHostFree(h_tmp2), "hipFree h_tmp2");
    checkCudaError(hipHostFree(h_y), "hipFree h_y");

    for (int i=0; i<NUM_STREAMS; ++i) {
        checkCudaError(hipStreamDestroy(streams[i]), "hipStreamDestroy streams");
    }
    checkCudaError(hipEventDestroy(start), "hipEventDestroy start");
    checkCudaError(hipEventDestroy(stop), "hipEventDestroy stop");
    checkCudaError(hipFree(d_x), "hipFree d_x");
    checkCudaError(hipFree(d_b), "hipFree d_b");
    checkCudaError(hipFree(d_tmp1), "hipFree d_tmp1");
    checkCudaError(hipFree(d_tmp2), "hipFree d_tmp2");
    checkCudaError(hipFree(d_y), "hipFree d_y");

    return 0;
}