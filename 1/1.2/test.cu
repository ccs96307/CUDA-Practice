#include <stdio.h>
#include <random>
#include <hip/hip_runtime.h>
#include <math.h>

#define N 20000000
#define NUM_THREADS 256
#define ITERS 200 // 增加迭代次數讓效果更明顯

#define TOTAL_DATA_SIZE (N * sizeof(float))
static void checkCudaError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// ===================================================================
// KERNEL 1: 純粹的分支發散
// if/else 中有不同的、耗時的迴圈
// ===================================================================
__global__ void true_divergeKernel(const float* in, float* out, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        float x = in[idx];
        float result = 0.0f;

        if (x > 0.0f) {
            // 路徑 A: 迴圈加 1.0
            for (int i = 0; i < ITERS; ++i) {
                result += 1.0f;
            }
        } else {
            // 路徑 B: 迴圈加 2.0
            for (int i = 0; i < ITERS; ++i) {
                result += 2.0f;
            }
        }
        out[idx] = result + x;
    }
}

// ===================================================================
// KERNEL 2: 真正無分支的優化
// 在迴圈外決定好參數，迴圈內部指令完全統一
// ===================================================================
__global__ void true_nonDivergeKernel(const float* in, float* out, int n) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < n) {
        float x = in[idx];
        
        // 關鍵：在迴圈開始前，用一個廉價的條件移動決定好增量
        float increment = (x > 0.0f) ? 1.0f : 2.0f;

        float result = 0.0f;
        
        // 這個迴圈對於Warp中的所有執行緒，指令都是100%相同的
        for (int i = 0; i < ITERS; ++i) {
            result += increment;
        }

        out[idx] = result + x;
    }
}

// Main function
int main() {
    printf("N = %d, ITERS = %d\n", N, ITERS);

    std::mt19937 gen(std::random_device{}());
    std::uniform_real_distribution<float> dist(-1.5f, 1.5f);

    float *h_x, *h_y;
    checkCudaError(hipHostMalloc((void**)&h_x, TOTAL_DATA_SIZE, hipHostMallocDefault), "hipHostMalloc h_x");
    checkCudaError(hipHostMalloc((void**)&h_y, TOTAL_DATA_SIZE, hipHostMallocDefault), "hipHostMalloc h_y");

    for (int i = 0; i < N; ++i) { h_x[i] = dist(gen); }

    float *d_x, *d_y;
    checkCudaError(hipMalloc((void**)&d_x, TOTAL_DATA_SIZE), "hipMalloc d_x");
    checkCudaError(hipMalloc((void**)&d_y, TOTAL_DATA_SIZE), "hipMalloc d_y");
    checkCudaError(hipMemcpy(d_x, h_x, TOTAL_DATA_SIZE, hipMemcpyHostToDevice), "hipMemcpy");

    hipEvent_t start, stop;
    checkCudaError(hipEventCreate(&start), "hipEventCreate start");
    checkCudaError(hipEventCreate(&stop), "hipEventCreate stop");

    dim3 block(NUM_THREADS);
    dim3 grid((N + NUM_THREADS - 1) / NUM_THREADS);
    float ms;

    // --- 測試 Divergence Kernel ---
    checkCudaError(hipEventRecord(start), "start");
    true_divergeKernel<<<grid, block>>>(d_x, d_y, N);
    checkCudaError(hipEventRecord(stop), "stop");
    checkCudaError(hipEventSynchronize(stop), "sync");
    checkCudaError(hipEventElapsedTime(&ms, start, stop), "elapsed");
    printf("True Divergence Kernel Time:   %f ms\n", ms);

    // --- 測試 Non-Divergence Kernel ---
    checkCudaError(hipEventRecord(start), "start");
    true_nonDivergeKernel<<<grid, block>>>(d_x, d_y, N);
    checkCudaError(hipEventRecord(stop), "stop");
    checkCudaError(hipEventSynchronize(stop), "sync");
    checkCudaError(hipEventElapsedTime(&ms, start, stop), "elapsed");
    printf("True Non-Divergence Kernel Time: %f ms\n", ms);

    // Cleanup
    hipFree(d_x); hipFree(d_y);
    hipHostFree(h_x); hipHostFree(h_y);
    hipEventDestroy(start); hipEventDestroy(stop);

    return 0;
}