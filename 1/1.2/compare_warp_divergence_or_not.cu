#include <stdio.h>
#include <random>
#include <hip/hip_runtime.h>

#define N 10000000
#define NUM_THREADS 256
#define NUM_STREAMS 4

#if N % NUM_STREAMS != 0
#error N must be divisible by NUM_STREAMS
#endif

#define CHUNK_SIZE (N / NUM_STREAMS)
#define CHUNK_BYTES (CHUNK_SIZE * sizeof(float))
#define TOTAL_DATA_SIZE (N * sizeof(float))

#define ITERS 100


__global__ void divergeKernel(const float* x1, float* y1, int chunkSize) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < chunkSize) {
        float x = x1[idx];
        float result;
        
        if (x > 0.5f) {
            result = x;
            for (int i=0; i<ITERS; ++i) {
                result = sinf(result) * 0.5f + cosf(result) * 0.5f;
            }
        }
        else {
            result = x;
        }

        y1[idx] = result;
    }
}


__global__ void nonDivergeKernel(const float* x2, float* y2, int chunkSize) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;

    if (idx < chunkSize) {
        float x = x2[idx];

        bool condition = (x > 0.5f);

        float heavy_result = x;
        for (int i=0; i<ITERS; ++i) {
            heavy_result = sinf(heavy_result) * 0.5f + cosf(heavy_result) * 0.5f;
        }

        float simple_result = x;

        y2[idx] = condition ? heavy_result : simple_result;
    }
}


static void checkCudaError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}


float getRand(std::mt19937& gen, std::uniform_real_distribution<float>& dist) {
    return dist(gen);
}


int main() {
    // Init random number generator and distribution
    std::mt19937 gen(std::random_device{}());
    std::uniform_real_distribution<float> dist(-1.5f, 1.5f);

    // Init host data (try to use fixed memory)
    float *h_x1, *h_x2, *h_y1, *h_y2;
    checkCudaError(hipHostMalloc((void**)&h_x1, TOTAL_DATA_SIZE, hipHostMallocDefault), "hipHostMalloc h_x1");
    checkCudaError(hipHostMalloc((void**)&h_x2, TOTAL_DATA_SIZE, hipHostMallocDefault), "hipHostMalloc h_x2");
    checkCudaError(hipHostMalloc((void**)&h_y1, TOTAL_DATA_SIZE, hipHostMallocDefault), "hipHostMalloc h_y1");
    checkCudaError(hipHostMalloc((void**)&h_y2, TOTAL_DATA_SIZE, hipHostMallocDefault), "hipHostMalloc h_y2");

    for (int i = 0; i < N; ++i) {
        h_x1[i] = getRand(gen, dist);
        h_x2[i] = h_x1[i];
    }

    // Init device data
    float *d_x1, *d_x2, *d_y1, *d_y2;
    checkCudaError(hipMalloc((void**)&d_x1, TOTAL_DATA_SIZE), "hipMalloc d_x1");
    checkCudaError(hipMalloc((void**)&d_x2, TOTAL_DATA_SIZE), "hipMalloc d_x2");
    checkCudaError(hipMalloc((void**)&d_y1, TOTAL_DATA_SIZE), "hipMalloc d_y1");
    checkCudaError(hipMalloc((void**)&d_y2, TOTAL_DATA_SIZE), "hipMalloc d_y2");

    // Copy data from host to device
    checkCudaError(hipMemcpyAsync(
        d_x1,
        h_x1, 
        CHUNK_BYTES * NUM_STREAMS,
        hipMemcpyHostToDevice
    ), "hipMemcpy h_x1 => d_x1");
    checkCudaError(hipMemcpyAsync(
        d_x2,
        h_x2, 
        CHUNK_BYTES * NUM_STREAMS,
        hipMemcpyHostToDevice
    ), "hipMemcpy h_x2 => d_x2");

    // Create Streams and Events
    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; ++i) {
        checkCudaError(hipStreamCreate(&streams[i]), "hipStreamCreate");
    }
    hipEvent_t start, stop;
    checkCudaError(hipEventCreate(&start), "hipEventCreate start");
    checkCudaError(hipEventCreate(&stop), "hipEventCreate stop");

    // Start to record
    checkCudaError(hipEventRecord(start), "hipEventRecord start");

    // Assign tasks to different streams
    for (int i=0; i<NUM_STREAMS; ++i) {
        int offset = i * CHUNK_SIZE;

        // Init block and grid
        dim3 block(NUM_THREADS);
        dim3 grid((CHUNK_SIZE + block.x - 1) / block.x);

        // Launch Add Kernel
        divergeKernel<<<grid, block, 0, streams[i]>>>(
            d_x1 + offset,
            d_y1 + offset,
            CHUNK_SIZE
        );
        checkCudaError(hipGetLastError(), "divergeKernel launched");
    }

    // Sync all operations
    checkCudaError(hipDeviceSynchronize(), "hipDeviceSynchronize for timing");

    // Stop to record
    checkCudaError(hipEventRecord(stop), "hipEventRecord stop");
    checkCudaError(hipEventSynchronize(stop), "hipEventSynchronize stop");

    // Print the time
    float ms = 0;
    checkCudaError(hipEventElapsedTime(&ms, start, stop), "hipEventElapsedTime");
    printf("Divergence Kernel Spend Time with %d streams: %f ms\n", NUM_STREAMS, ms);

    // Start to record
    checkCudaError(hipEventRecord(start), "hipEventRecord start");

    // Assign tasks to different streams
    for (int i=0; i<NUM_STREAMS; ++i) {
        int offset = i * CHUNK_SIZE;

        // Init block and grid
        dim3 block(NUM_THREADS);
        dim3 grid((CHUNK_SIZE + block.x - 1) / block.x);

        // Launch Add Kernel
        nonDivergeKernel<<<grid, block, 0, streams[i]>>>(
            d_x2 + offset,
            d_y2 + offset,
            CHUNK_SIZE
        );
        checkCudaError(hipGetLastError(), "nonDivergeKernel launched");
    }

    // Sync all operations
    checkCudaError(hipDeviceSynchronize(), "hipDeviceSynchronize for timing");

    // Stop to record
    checkCudaError(hipEventRecord(stop), "hipEventRecord stop");
    checkCudaError(hipEventSynchronize(stop), "hipEventSynchronize stop");

    // Print the time
    ms = 0;
    checkCudaError(hipEventElapsedTime(&ms, start, stop), "hipEventElapsedTime");
    printf("Non-Divergence Kernel Spend Time with %d streams: %f ms\n", NUM_STREAMS, ms);

    // Destory
    checkCudaError(hipHostFree(h_x1), "hipFree h_x1");
    checkCudaError(hipHostFree(h_x2), "hipFree h_x2");
    checkCudaError(hipHostFree(h_y1), "hipFree h_y1");
    checkCudaError(hipHostFree(h_y2), "hipFree h_y2");

    for (int i=0; i<NUM_STREAMS; ++i) {
        checkCudaError(hipStreamDestroy(streams[i]), "hipStreamDestroy streams");
    }
    checkCudaError(hipEventDestroy(start), "hipEventDestroy start");
    checkCudaError(hipEventDestroy(stop), "hipEventDestroy stop");
    checkCudaError(hipFree(d_x1), "hipFree d_x1");
    checkCudaError(hipFree(d_x2), "hipFree d_x2");
    checkCudaError(hipFree(d_y1), "hipFree d_y1");
    checkCudaError(hipFree(d_y2), "hipFree d_y2");

    return 0;
}