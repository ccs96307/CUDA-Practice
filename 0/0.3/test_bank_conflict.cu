#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>


#define N 32
#define ITERATIONS 1000


__global__ void withoutPaddingKernel(float *out) {
    __shared__ float tile[N][N];  // No padding
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    float sum = 0.0f;
    for (int i=0; i<ITERATIONS; ++i) {
        tile[tx][ty] = tx * 1.0f;
        __syncthreads();
        sum += tile[ty][tx]; // Column-wise access (bad)
        __syncthreads();
    }
    out[tx * N + ty] = sum;
}


__global__ void withPaddingKernel(float *out) {
    __shared__ float tile[N][N+1];  // Padding by 1
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    float sum = 0.0f;
    for (int i=0; i<ITERATIONS; ++i) {
        tile[tx][ty] = tx * 1.0f;
        __syncthreads();
        sum += tile[ty][tx]; // Column-wise access (good)
        __syncthreads();
    }
    out[tx * N + ty] = sum;
}


static void checkCudaError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}


void benchmark(void (*kernel)(float*), float* d_out, const char* name) {
    hipEvent_t start, stop;
    checkCudaError(hipEventCreate(&start), "hipEventCreate start");
    checkCudaError(hipEventCreate(&stop), "hipEventCreate stop");

    dim3 block(N, N);
    dim3 grid(1, 1);

    checkCudaError(hipEventRecord(start), "hipEventRecord start");
    kernel<<<grid, block>>>(d_out);
    checkCudaError(hipEventRecord(stop), "hipEventRecord stop");

    checkCudaError(hipEventSynchronize(stop), "hipEventSynchronize stop");
    float ms = 0;
    checkCudaError(hipEventElapsedTime(&ms, start, stop), "cudaEventElapsedTime: stop - start and assign to `ms`");
    std::cout << name << " time: " << ms << " ms\n";

    checkCudaError(hipEventDestroy(start), "hipEventDestroy start");
    checkCudaError(hipEventDestroy(stop), "hipEventDestroy stop");
}


int main() {
    float *d_out;
    checkCudaError(hipMalloc(&d_out, sizeof(float) * N * N), "hipMalloc d_out");

    benchmark(withPaddingKernel, d_out, "With Padding");
    benchmark(withoutPaddingKernel, d_out, "Without Padding");
    benchmark(withPaddingKernel, d_out, "With Padding");

    checkCudaError(hipFree(d_out), "hipFree d_out");

    return 0;
}