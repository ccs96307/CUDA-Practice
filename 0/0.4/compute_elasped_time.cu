#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 32
#define DATA_SIZE (N * sizeof(float))


__global__ void computeKernel(float *d_B) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        d_B[idx] += idx;
    }
}


static void checkCudaError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}


int main() {
    // Allocate host memory
    float* h_A = new float[N];
    for (int i=0; i<N; ++i) {
        h_A[i] = 1.0;
    }

    // Allocate device memroy
    float *d_A;
    checkCudaError(hipMalloc((void**)&d_A, DATA_SIZE), "hipMalloc d_A");

    // Use cudaStream_t to startup the kernel
    hipStream_t stream1;
    checkCudaError(hipStreamCreate(&stream1), "hipStreamCreate stream1");

    // Copy data from host to device
    checkCudaError(hipMemcpyAsync(d_A, h_A, DATA_SIZE, hipMemcpyHostToDevice), "hipMemcpyAsync d_A to stream1");
 
    // Count time
    hipEvent_t start, stop;
    checkCudaError(hipEventCreate(&start), "hipEventCreate start");
    checkCudaError(hipEventCreate(&stop), "hipEventCreate stop");

    // Launch kernel (stream1)
    checkCudaError(hipEventRecord(start, stream1), "hipEventRecord start");

    computeKernel<<<1, N, 0, stream1>>>(d_A);
    checkCudaError(hipGetLastError(), "computeKernel launch");
    printf("computeKernel launched in stream1\n");

    checkCudaError(hipEventRecord(stop, stream1), "hipEventRecord stop");
    checkCudaError(hipEventSynchronize(stop), "hipEventSynchronize stop");

    float milliseconds = 0;
    checkCudaError(hipEventElapsedTime(&milliseconds, start, stop), "hipEventElapsedTime stream1");
    printf("GPU kernel (stream1) time: %f ms\n", milliseconds);

    // Copy result from device to host
    checkCudaError(hipMemcpyAsync(h_A, d_A, DATA_SIZE, hipMemcpyDeviceToHost), "hipMemcpyAsync d_A to host from stream1");

    // Sync to make sure kernel is done
    checkCudaError(hipDeviceSynchronize(), "cudaDeviceSychronize");

    for (int i=0; i<N; ++i) {
        printf("h_A[%d] = %f\n", i, h_A[i]);
    }

    // Destroy
    delete[] h_A;
    checkCudaError(hipStreamDestroy(stream1), "hipStreamDestroy stream1");
    checkCudaError(hipEventDestroy(start), "hipEventDestroy start");
    checkCudaError(hipEventDestroy(stop), "hipEventDestroy stop");
    checkCudaError(hipFree(d_A), "hipFree d_A");

    return 0;
}