#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 10000000
#define NUM_THREADS 256
#define NUM_STREAMS 16

#if N % NUM_STREAMS != 0
#error N must be divisible by NUM_STREAMS
#endif

#define CHUNK_SIZE (N / NUM_STREAMS)
#define CHUNK_BYTES (CHUNK_SIZE * sizeof(float))
#define TOTAL_DATA_SIZE (N * sizeof(float))


__global__ void addKernel(float *a, float *b, float *out, int chunkSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < chunkSize) {
        float temp_a = a[idx];
        float temp_b = b[idx];
        for (int i = 0; i < 1000; ++i) {
            temp_a = temp_a * 0.9999f + temp_b * 0.0001f;
        }
        out[idx] = temp_a;
    }
}


static void checkCudaError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}


int main() {
    float *h_a, *h_b, *h_out;
    checkCudaError(hipHostMalloc((void**)&h_a, TOTAL_DATA_SIZE, hipHostMallocDefault), "hipHostMalloc h_a");
    checkCudaError(hipHostMalloc((void**)&h_b, TOTAL_DATA_SIZE, hipHostMallocDefault), "hipHostMalloc h_b");
    checkCudaError(hipHostMalloc((void**)&h_out, TOTAL_DATA_SIZE, hipHostMallocDefault), "hipHostMalloc h_out");

    for (int i = 0; i < N; ++i) {
        h_a[i] = static_cast<float>(i);
        h_b[i] = static_cast<float>(i);
    }

    float *d_a, *d_b, *d_out;
    checkCudaError(hipMalloc((void**)&d_a, TOTAL_DATA_SIZE), "hipMalloc d_a");
    checkCudaError(hipMalloc((void**)&d_b, TOTAL_DATA_SIZE), "hipMalloc d_b");
    checkCudaError(hipMalloc((void**)&d_out, TOTAL_DATA_SIZE), "hipMalloc d_out");

    hipStream_t streams[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; ++i) {
        checkCudaError(hipStreamCreate(&streams[i]), "hipStreamCreate");
    }

    hipEvent_t start, stop;
    checkCudaError(hipEventCreate(&start), "hipEventCreate start");
    checkCudaError(hipEventCreate(&stop), "hipEventCreate stop");

    // === CUDA Graph Capture ===
    hipGraph_t graph;
    hipGraphExec_t instance;

    checkCudaError(hipStreamBeginCapture(streams[0], hipStreamCaptureModeGlobal), "Begin graph capture");

    for (int i = 0; i < NUM_STREAMS; ++i) {
        int offset = i * CHUNK_SIZE;

        // Copy to device
        checkCudaError(hipMemcpyAsync(d_a + offset, h_a + offset, CHUNK_BYTES, hipMemcpyHostToDevice, streams[0]), "MemcpyAsync h_a");
        checkCudaError(hipMemcpyAsync(d_b + offset, h_b + offset, CHUNK_BYTES, hipMemcpyHostToDevice, streams[0]), "MemcpyAsync h_b");

        // Launch kernel
        dim3 block(NUM_THREADS);
        dim3 grid((CHUNK_SIZE + block.x - 1) / block.x);
        addKernel<<<grid, block, 0, streams[0]>>>(d_a + offset, d_b + offset, d_out + offset, CHUNK_SIZE);
        checkCudaError(hipGetLastError(), "addKernel launch");

        // Copy back
        checkCudaError(hipMemcpyAsync(h_out + offset, d_out + offset, CHUNK_BYTES, hipMemcpyDeviceToHost, streams[0]), "MemcpyAsync d_out");
    }

    checkCudaError(hipStreamEndCapture(streams[0], &graph), "End graph capture");
    checkCudaError(hipGraphInstantiate(&instance, graph, NULL, NULL, 0), "Graph instantiate");

    // Timing
    checkCudaError(hipEventRecord(start), "hipEventRecord start");
    checkCudaError(hipGraphLaunch(instance, streams[0]), "Graph launch");
    checkCudaError(hipStreamSynchronize(streams[0]), "Stream synchronize");
    checkCudaError(hipEventRecord(stop), "hipEventRecord stop");
    checkCudaError(hipEventSynchronize(stop), "hipEventSynchronize stop");

    float ms = 0;
    checkCudaError(hipEventElapsedTime(&ms, start, stop), "Elapsed time");
    printf("GPU Kernel Spend Time with %d streams + CUDA Graph: %f ms\n", NUM_STREAMS, ms);

    // Cleanup
    checkCudaError(hipHostFree(h_a), "hipHostFree h_a");
    checkCudaError(hipHostFree(h_b), "hipHostFree h_b");
    checkCudaError(hipHostFree(h_out), "hipHostFree h_out");

    for (int i = 0; i < NUM_STREAMS; ++i) {
        checkCudaError(hipStreamDestroy(streams[i]), "hipStreamDestroy");
    }

    checkCudaError(hipEventDestroy(start), "hipEventDestroy start");
    checkCudaError(hipEventDestroy(stop), "hipEventDestroy stop");

    checkCudaError(hipFree(d_a), "hipFree d_a");
    checkCudaError(hipFree(d_b), "hipFree d_b");
    checkCudaError(hipFree(d_out), "hipFree d_out");

    checkCudaError(hipGraphDestroy(graph), "hipGraphDestroy");
    checkCudaError(hipGraphExecDestroy(instance), "hipGraphExecDestroy");

    return 0;
}
