#include<iostream>
#include<random>
#include<hip/hip_runtime.h>

#define N 32
#define C 4
#define H 256
#define W 512

#define TILE_H 8
#define TILE_W 32


__launch_bounds__(256)
__global__ void nchw_to_nhwc_kernel(const float* __restrict__ input, 
                                    float* __restrict__ output) {
    int n = blockIdx.z;
    int h = blockIdx.y * blockDim.y + threadIdx.y;
    int w = blockIdx.x * blockDim.x + threadIdx.x;

    if (h >= H || w >= W) {
        return;
    }

    #pragma unroll
    for (int c=0; c<C; ++c) {
        int inputIdx = ((n * C + c) * H + h) * W + w;
        int outputIdx = ((n * H + h) * W + w) * C + c;

        output[outputIdx] = input[inputIdx];
    }
}


static void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}


float genRand(std::mt19937& gen, std::uniform_real_distribution<float>& dist) {
    return dist(gen);
}


int main() {
    // Init
    int dataNum = N * C * H * W;
    int dataSize = dataNum * sizeof(float);
    float *h_input, *h_output;

    // Malloc host memory
    checkCudaError(hipHostMalloc((void**)&h_input, dataSize, hipHostMallocDefault), "hipHostMalloc h_input");
    checkCudaError(hipHostMalloc((void**)&h_output, dataSize, hipHostMallocDefault), "hipHostMalloc h_output");

    // Random initialization
    // Init random number generator and distribution
    std::mt19937 gen(std::random_device{}());
    std::uniform_real_distribution<float> dist(-1.5f, 1.5f);

    for (int i=0; i<dataNum; ++i) {
        h_input[i] = float(genRand(gen, dist));
    }

    // Init device data
    float *d_input, *d_output;
    checkCudaError(hipMalloc((void**)&d_input, dataSize), "hipMalloc d_input");
    checkCudaError(hipMalloc((void**)&d_output, dataSize), "hipMalloc d_output");

    // Create streams and Events
    hipStream_t stream;
    checkCudaError(hipStreamCreate(&stream), "hipStreamCreate stream");

    hipEvent_t start, stop;
    checkCudaError(hipEventCreate(&start), "hipEventCreate start");
    checkCudaError(hipEventCreate(&stop), "hipEventCreate stop");

    // Copy data
    checkCudaError(hipMemcpy(d_input, h_input, dataSize, hipMemcpyHostToDevice), "hipMemcpy h_input => d_input");
    
    // Start to record
    checkCudaError(hipEventRecord(start), "hipEventRecord start");
    

    // Launch kernel
    dim3 block(TILE_W, TILE_H);
    dim3 grid((W + TILE_W - 1) / TILE_W,
              (H + TILE_H - 1) / TILE_H,
              N);
    nchw_to_nhwc_kernel<<<grid, block>>>(d_input, d_output);

    // Stop to record
    checkCudaError(hipEventRecord(stop), "hipEventRecord stop");
    checkCudaError(hipEventSynchronize(stop), "hipEventSynchronize stop");

    // Print timing
    float ms = 0.0f;
    checkCudaError(hipEventElapsedTime(&ms, start, stop), "hipEventElapsedTime");
    printf("Optimal Kernel Spend Time: %f ms\n", ms);
    
    // Copy data
    checkCudaError(hipMemcpy(h_output, d_output, dataSize, hipMemcpyDeviceToHost), "hipMemcpy d_output => h_output");
    
    // Destroy
    checkCudaError(hipHostFree(h_input), "hipHostFree h_input");
    checkCudaError(hipHostFree(h_output), "hipHostFree h_output");
    checkCudaError(hipStreamDestroy(stream), "hipStreamDestroy stream");
    checkCudaError(hipEventDestroy(start), "hipEventDestroy start");
    checkCudaError(hipEventDestroy(stop), "hipEventDestroy stop");
    checkCudaError(hipFree(d_input), "hipFree d_input");
    checkCudaError(hipFree(d_output), "hipFree d_output");
    return 0;
}