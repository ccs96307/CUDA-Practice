#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 320000000
#define DATA_SIZE (N * sizeof(float))


__global__ void alignedKernel(float *d_A) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        float val = d_A[idx];
        d_A[idx] = val * 2.0f;
    }
}


__global__ void stridedKernel(float *d_B) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        float val = d_B[idx * 2 % N];
        d_B[idx] = val * 2.0f;
    }
}


__global__ void randomKernel(float *d_C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        int randIdx = (idx * 2 + 13) % N;
        float val = d_C[randIdx];
        d_C[randIdx] = val * 2.0f;
    }
}


static void checkCudaError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}


void benchmark(void (*kernel)(float*), float *d_out, const char *name) {
    hipEvent_t start, stop;
    checkCudaError(hipEventCreate(&start), "hipEventCreate &start");
    checkCudaError(hipEventCreate(&stop), "hipEventCreate &stop");

    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;
    dim3 block(blockSize);
    dim3 grid(gridSize);

    checkCudaError(hipEventRecord(start), "hipEventRecord start");
    kernel<<<grid, block>>>(d_out);
    checkCudaError(hipGetLastError(), "Kernel Launch");
    checkCudaError(hipDeviceSynchronize(), "hipDeviceSynchronize after kernel");
    checkCudaError(hipEventRecord(stop), "hipEventRecord stop");
    checkCudaError(hipEventSynchronize(stop), "hipEventSynchronize stop");

    // Show time
    float ms = 0;
    checkCudaError(hipEventElapsedTime(&ms, start, stop), "hipEventElapsedTime");
    printf("%s time: %f sec\n", name, ms / 1000);

    checkCudaError(hipEventDestroy(start), "hipEventDestroy start");
    checkCudaError(hipEventDestroy(stop), "hipEventDestroy stop");    
}


int main() {
    // Init
    float *h_A = new float[N];
    float *h_B = new float[N];
    float *h_C = new float[N];
    for (int i=0; i<N; ++i) {
        h_A[i] = 1.0 * (i + 1);
        h_B[i] = 1.0 * (i + 1);
        h_C[i] = 1.0 * (i + 1);        
    }

    // Allocate device memory
    float *d_A, *d_B, *d_C;
    checkCudaError(hipMalloc((void**)&d_A, DATA_SIZE), "hipMalloc d_A");
    checkCudaError(hipMalloc((void**)&d_B, DATA_SIZE), "hipMalloc d_B");
    checkCudaError(hipMalloc((void**)&d_C, DATA_SIZE), "hipMalloc d_C");

    // Memory copy
    checkCudaError(hipMemcpy(d_A, h_A, DATA_SIZE, hipMemcpyHostToDevice), "hipMemcpy h_A to d_A");
    checkCudaError(hipMemcpy(d_B, h_B, DATA_SIZE, hipMemcpyHostToDevice), "hipMemcpy h_B to d_B");
    checkCudaError(hipMemcpy(d_C, h_C, DATA_SIZE, hipMemcpyHostToDevice), "hipMemcpy h_C to d_C");

    // Benchmark
    benchmark(alignedKernel, d_A, "alignedKernel");
    benchmark(stridedKernel, d_B, "stridedKernel");
    benchmark(randomKernel, d_C, "randomKernel");

    // Free the memory
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    checkCudaError(hipFree(d_A), "hipFree d_A");
    checkCudaError(hipFree(d_B), "hipFree d_B");
    checkCudaError(hipFree(d_C), "hipFree d_C");

    return 0;
}
