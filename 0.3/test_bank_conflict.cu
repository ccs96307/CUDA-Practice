#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>


#define N 32
#define ITERATIONS 1000


__global__ void withoutPaddingKernel(float *out) {
    __shared__ float tile[N][N];  // No padding
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    float sum = 0.0f;
    for (int i=0; i<ITERATIONS; ++i) {
        tile[tx][ty] = tx * 1.0f;
        __syncthreads();
        sum += tile[ty][tx]; // Column-wise access (bad)
        __syncthreads();
    }
    out[tx * N + ty] = sum;
}


__global__ void withPaddingKernel(float *out) {
    __shared__ float tile[N][N+1];  // Padding by 1
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    float sum = 0.0f;
    for (int i=0; i<ITERATIONS; ++i) {
        tile[tx][ty] = tx * 1.0f;
        __syncthreads();
        sum += tile[ty][tx]; // Column-wise access (good)
        __syncthreads();
    }
    out[tx * N + ty] = sum;
}


void benchmark(void (*kernel)(float*), float* d_out, const char* name) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    dim3 block(N, N);
    dim3 grid(1, 1);

    hipEventRecord(start);
    kernel<<<grid, block>>>(d_out);
    hipEventRecord(stop);

    hipEventSynchronize(stop);
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    std::cout << name << " time: " << ms << " ms\n";

    hipEventDestroy(start);
    hipEventDestroy(stop);
}


int main() {
    float *d_out;
    hipMalloc(&d_out, sizeof(float) * N * N);

    benchmark(withoutPaddingKernel, d_out, "Without Padding");
    benchmark(withPaddingKernel, d_out, "With Padding");
    
    hipFree(d_out);

    return 0;
}