#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 512
#define DATA_SIZE (N * sizeof(int))


__global__ void squareKernel(int *data) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        data[idx] = data[idx] * data[idx];
    }
}


static void checkCudaError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}


int main() {
    // Init
    int *h_data = new int[N];
    for (int i=0; i<N; ++i) {
        h_data[i] = i;
    }

    // Allocate device data
    int *d_data;
    checkCudaError(hipMalloc((void**)&d_data, DATA_SIZE), "hipMalloc d_data");

    // Copy (host to device)
    checkCudaError(hipMemcpy(d_data, h_data, DATA_SIZE, hipMemcpyHostToDevice), "hipMemcpy h_data into d_data");

    // Launch kernel
    dim3 block(32);
    dim3 grid((N + block.x - 1) / block.x);
    squareKernel<<<grid, block>>>(d_data);
    checkCudaError(hipGetLastError(), "Kernel launch");

    // Sync to make sure kernel is done
    checkCudaError(hipDeviceSynchronize(), "cudaDeviceSychronize");

    // Copy (device to host)
    checkCudaError(hipMemcpy(h_data, d_data, DATA_SIZE, hipMemcpyDeviceToHost), "hipMemcpy d_data into h_data");
    
    // Print the first 10 data
    for (int i=0; i<10; ++i) {
        printf("Results: %d^2 = %d\n", i, h_data[i]);
    }

    // Release
    delete[] h_data;
    checkCudaError(hipFree(d_data), "hipFree d_data");

    return 0;
}




