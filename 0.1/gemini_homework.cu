#include <stdio.h>
#include <hip/hip_runtime.h>


#define N 10000


__global__ void square(float *A) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < N) {
        A[i] = A[i] * A[i];
    }
}


int main() {
    hipError_t err = hipSuccess;

    // Init
    size_t size = N * sizeof(float);

    // Allocate the host input vector
    float *h_A = (float *)malloc(size);

    // Initialize the host input vectors
    for (int i=0; i<N; ++i) {
        h_A[i] = rand();
    }

    // Allocate the device input vectors
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy into GPU
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock; // 40 * 256 = 10240
    square<<<blocksPerGrid, threadsPerBlock>>>(d_A);
    err = hipGetLastError();

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Sync to make sure kernel is done
    hipDeviceSynchronize();

    // Copy result from device to host
    hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Failed to copy vector A from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Print
    for (int i=0; i<N; ++i) {
        printf("h_A[%d] = %.4f\n", i, h_A[i]);
    }

    // Free device memory
    free(h_A);
    hipFree(d_A);

    return 0;
}