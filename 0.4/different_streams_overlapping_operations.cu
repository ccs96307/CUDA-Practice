#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 10000000
#define NUM_THREADS 256
#define NUM_STREAMS 4

// Check N can be divisible
#if N % NUM_STREAMS != 0
#error N must be divisible by NUM_STREAMS
#endif

#define CHUNK_SIZE (N / NUM_STREAMS)
#define CHUNK_BYTES (CHUNK_SIZE * sizeof(float))
#define TOTAL_DATA_SIZE (N * sizeof(float))


__global__ void addKernel(float *a, float *b, float *out, int chunkSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < chunkSize) {
        // Increase many operations
        float temp_a = a[idx];
        float temp_b = b[idx];

        for (int i=0; i<1000; ++i) {
            temp_a = temp_a * 0.9999f + temp_b * 0.0001f;
        }

        out[idx] = temp_a;
    }
}


static void checkCudaError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}


int main() {
    // Init host data (try to use fixed memory)
    float *h_a, *h_b, *h_out;
    checkCudaError(hipHostMalloc((void**)&h_a, TOTAL_DATA_SIZE, hipHostMallocDefault), "hipHostMalloc h_a");
    checkCudaError(hipHostMalloc((void**)&h_b, TOTAL_DATA_SIZE, hipHostMallocDefault), "hipHostMalloc h_b");
    checkCudaError(hipHostMalloc((void**)&h_out, TOTAL_DATA_SIZE, hipHostMallocDefault), "hipHostMalloc h_out");

    for (int i=0; i<N; ++i) {
        h_a[i] = static_cast<float>(i);
        h_b[i] = static_cast<float>(i);
    }

    // Init device data
    float *d_a, *d_b, *d_out;
    checkCudaError(hipMalloc((void**)&d_a, TOTAL_DATA_SIZE), "hipMalloc d_a");
    checkCudaError(hipMalloc((void**)&d_b, TOTAL_DATA_SIZE), "hipMalloc d_b");
    checkCudaError(hipMalloc((void**)&d_out, TOTAL_DATA_SIZE), "hipMalloc d_out");

    // Create Streams and Events
    hipStream_t streams[NUM_STREAMS];
    for (int i=0; i<NUM_STREAMS; ++i) {
        checkCudaError(hipStreamCreate(&streams[i]), "hipStreamCreate");
    }
    hipEvent_t start, stop;
    checkCudaError(hipEventCreate(&start), "hipEventCreate start");
    checkCudaError(hipEventCreate(&stop), "hipEventCreate stop");

    // Start to record
    checkCudaError(hipEventRecord(start), "hipEventRecord start");

    // Assign tasks to different streams
    for (int i=0; i<NUM_STREAMS; ++i) {
        int offset = i * CHUNK_SIZE;

        // Copy data from host to device
        checkCudaError(hipMemcpyAsync(d_a + offset, h_a + offset, CHUNK_BYTES, hipMemcpyHostToDevice, streams[i]), "hipMemcpy h_a => d_a");
        checkCudaError(hipMemcpyAsync(d_b + offset, h_b + offset, CHUNK_BYTES, hipMemcpyHostToDevice, streams[i]), "hipMemcpy h_b => d_b");

        // Launch Kernel
        dim3 block(NUM_THREADS);
        dim3 grid((CHUNK_SIZE + block.x - 1) / block.x);
        addKernel<<<grid, block, 0, streams[i]>>>(
            d_a + offset,
            d_b + offset,
            d_out + offset,
            CHUNK_SIZE
        );

        checkCudaError(hipGetLastError(), "addKerenl launched");

        // Copy result from device to host
        checkCudaError(hipMemcpyAsync(h_out + offset, d_out + offset, CHUNK_BYTES, hipMemcpyDeviceToHost, streams[i]), "hipMemcpyAsync d_out to host");
    }

    // Sync all operations
    checkCudaError(hipDeviceSynchronize(), "hipDeviceSynchronize for timing");

    // Stop to record
    checkCudaError(hipEventRecord(stop), "hipEventRecord stop");
    checkCudaError(hipEventSynchronize(stop), "hipEventSynchronize stop");

    // Print the time
    float ms = 0;
    checkCudaError(hipEventElapsedTime(&ms, start, stop), "hipEventElapsedTime");
    printf("GPU Kernel Spend Time with %d streams: %f ms\n", NUM_STREAMS, ms);

    // Destory
    checkCudaError(hipHostFree(h_a), "hipFree h_a");
    checkCudaError(hipHostFree(h_b), "hipFree h_b");
    checkCudaError(hipHostFree(h_out), "hipFree h_out");

    for (int i=0; i<NUM_STREAMS; ++i) {
        checkCudaError(hipStreamDestroy(streams[i]), "hipStreamDestroy streams");
    }
    checkCudaError(hipEventDestroy(start), "hipEventDestroy start");
    checkCudaError(hipEventDestroy(stop), "hipEventDestroy stop");
    checkCudaError(hipFree(d_a), "hipFree d_a");
    checkCudaError(hipFree(d_b), "hipFree d_b");
    checkCudaError(hipFree(d_out), "hipFree d_out");
    
    return 0;
}