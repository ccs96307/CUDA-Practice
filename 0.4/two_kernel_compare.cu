#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 32
#define DATA_SIZE (N * sizeof(float))

__global__ void streamKernel1(float *d_A) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        d_A[idx] += idx;
    }
}


__global__ void streamKernel2(float *d_B) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        d_B[idx] += idx + 1;
    }
}


static void checkCudaError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}


int main() {
    // Allocate host memory
    float* h_A = new float[N];
    float* h_B = new float[N];
    for (int i=0; i<N; ++i) {
        h_A[i] = 1.0;
        h_B[i] = 1.0;
    }

    // Allocate device memroy
    float *d_A, *d_B;
    checkCudaError(hipMalloc((void**)&d_A, DATA_SIZE), "hipMalloc d_A");
    checkCudaError(hipMalloc((void**)&d_B, DATA_SIZE), "hipMalloc d_B");

    // Use cudaStream_t to startup two kernel in the same time
    hipStream_t stream1, stream2;
    checkCudaError(hipStreamCreate(&stream1), "hipStreamCreate stream1");
    checkCudaError(hipStreamCreate(&stream2), "hipStreamCreate stream2");

    printf("Streams created: stream1=%p, stream2=%p\n", (void*)stream1, (void*)stream2);

    // Copy data from host to device
    checkCudaError(hipMemcpyAsync(d_A, h_A, DATA_SIZE, hipMemcpyHostToDevice), "hipMemcpyAsync d_A to stream1");
    checkCudaError(hipMemcpyAsync(d_B, h_B, DATA_SIZE, hipMemcpyHostToDevice), "hipMemcpyAsync d_B to stream2");
 
    // Count time
    hipEvent_t start, stop;
    checkCudaError(hipEventCreate(&start), "hipEventCreate start");
    checkCudaError(hipEventCreate(&stop), "hipEventCreate stop");

    // Launch kernel (stream1)
    checkCudaError(hipEventRecord(start, stream1), "hipEventRecord start");

    streamKernel1<<<1, N, 0, stream1>>>(d_A);
    checkCudaError(hipGetLastError(), "streamKernel1 launch");
    printf("streamKernel1 launched in stream1\n");

    checkCudaError(hipEventRecord(stop, stream1), "hipEventRecord stop");
    checkCudaError(hipEventSynchronize(stop), "hipEventSynchronize stop");

    float milliseconds = 0;
    checkCudaError(hipEventElapsedTime(&milliseconds, start, stop), "hipEventElapsedTime stream1");
    printf("GPU kernel (stream2) time: %f ms\n", milliseconds);

    // Count time
    // cudaEvent_t start, stop;
    // checkCudaError(cudaEventCreate(&start), "cudaEventCreate start");
    // checkCudaError(cudaEventCreate(&stop), "cudaEventCreate stop");

    // Launch kernel (stream2)
    checkCudaError(hipEventRecord(start, stream2), "hipEventRecord start");

    streamKernel2<<<1, N, 0, stream2>>>(d_B);
    checkCudaError(hipGetLastError(), "streamKernel2 launch");
    printf("streamKernel2 launched in stream2\n");

    checkCudaError(hipEventRecord(stop, stream2), "hipEventRecord stop");
    checkCudaError(hipEventSynchronize(stop), "hipEventSynchronize stop");

    milliseconds = 0;
    checkCudaError(hipEventElapsedTime(&milliseconds, start, stop), "hipEventElapsedTime stream2");
    printf("GPU kernel (stream2) time: %f ms\n", milliseconds);

    // Copy result from device to host
    checkCudaError(hipMemcpyAsync(h_A, d_A, DATA_SIZE, hipMemcpyDeviceToHost), "hipMemcpyAsync d_A to host from stream1");
    checkCudaError(hipMemcpyAsync(h_B, d_B, DATA_SIZE, hipMemcpyDeviceToHost), "hipMemcpyAsync d_B to host from stream2");

    // Sync to make sure kernel is done
    checkCudaError(hipDeviceSynchronize(), "cudaDeviceSychronize");

    for (int i=0; i<N; ++i) {
        printf("h_A[%d] = %f\n", i, h_A[i]);
    }

    for (int i=0; i<N; ++i) {
        printf("h_B[%d] = %f\n", i, h_B[i]);
    }

    // Destroy streams
    checkCudaError(hipStreamDestroy(stream1), "hipStreamDestroy stream1");
    checkCudaError(hipStreamDestroy(stream2), "hipStreamDestroy stream2");
    

    delete[] h_A;
    delete[] h_B;
    checkCudaError(hipFree(d_A), "hipFree d_A");
    checkCudaError(hipFree(d_B), "hipFree d_B");

    return 0;
}