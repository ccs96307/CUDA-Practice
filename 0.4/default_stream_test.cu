#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 10000000
#define THREADS 256
#define DATA_SIZE (N * sizeof(float))



__global__ void addKernel(float *a, float *b, float *out) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N) {
        // Increase many operations
        float temp_a = a[idx];
        float temp_b = b[idx];

        for (int i=0; i<1000; ++i) {
            temp_a = temp_a * 0.9999f + temp_b * 0.0001f;
        }

        out[idx] = temp_a;
    }
}


static void checkCudaError(hipError_t err, const char *msg) {
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s: %s\n", msg, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

int main() {
    // Init host data
    float *h_a = new float[N];
    float *h_b = new float[N];
    float *h_out = new float[N];

    for (int i=0; i<N; ++i) {
        h_a[i] = i;
        h_b[i] = i;
    }

    // Init device data
    float *d_a, *d_b, *d_out;
    checkCudaError(hipMalloc((void**)&d_a, DATA_SIZE), "hipMalloc d_a");
    checkCudaError(hipMalloc((void**)&d_b, DATA_SIZE), "hipMalloc d_b");
    checkCudaError(hipMalloc((void**)&d_out, DATA_SIZE), "hipMalloc d_out");

    // Init Event
    hipEvent_t start, stop;
    checkCudaError(hipEventCreate(&start), "hipEventCreate start");
    checkCudaError(hipEventCreate(&stop), "hipEventCreate stop");

    // Start to record
    checkCudaError(hipEventRecord(start), "hipEventRecord start");

    // Copy data from host to device
    checkCudaError(hipMemcpy(d_a, h_a, DATA_SIZE, hipMemcpyHostToDevice), "hipMemcpy h_a => d_a");
    checkCudaError(hipMemcpy(d_b, h_b, DATA_SIZE, hipMemcpyHostToDevice), "hipMemcpy h_b => d_b");
    checkCudaError(hipMemcpy(d_out, h_out, DATA_SIZE, hipMemcpyHostToDevice), "hipMemcpy h_out => d_out");

    // Launch Kernel
    dim3 block(THREADS);
    dim3 grid((N + block.x - 1) / block.x);
    addKernel<<<grid, block>>>(d_a, d_b, d_out);

    checkCudaError(hipGetLastError(), "addKerenl launched");

    // Stop to record
    checkCudaError(hipEventRecord(stop), "hipEventRecord stop");
    checkCudaError(hipEventSynchronize(stop), "hipEventSynchronize stop");

    // Print the time
    float ms = 0;
    checkCudaError(hipEventElapsedTime(&ms, start, stop), "hipEventElapsedTime");
    printf("GPU Kernel Spend Time: %f ms\n", ms);

    // Copy result from device to host
    checkCudaError(hipMemcpyAsync(h_out, d_out, DATA_SIZE, hipMemcpyDeviceToHost), "hipMemcpyAsync d_out to host");

    // Destory
    delete[] h_a;
    delete[] h_b;
    delete[] h_out;
    checkCudaError(hipEventDestroy(start), "hipEventDestroy start");
    checkCudaError(hipEventDestroy(stop), "hipEventDestroy stop");
    checkCudaError(hipFree(d_a), "hipFree d_a");
    checkCudaError(hipFree(d_b), "hipFree d_b");
    checkCudaError(hipFree(d_out), "hipFree d_out");
    
    return 0;
}